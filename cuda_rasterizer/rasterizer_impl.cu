#include "hip/hip_runtime.h"
/*
 * The original code is under the following copyright:
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE_GS.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 * 
 * The modifications of the code are under the following copyright:
 * Copyright (C) 2024, University of Liege, KAUST and University of Oxford
 * TELIM research group, http://www.telecom.ulg.ac.be/
 * IVUL research group, https://ivul.kaust.edu.sa/
 * VGG research group, https://www.robots.ox.ac.uk/~vgg/
 * All rights reserved.
 * The modifications are under the LICENSE.md file.
 *
 * For inquiries contact jan.held@uliege.be
 */

 #include "rasterizer_impl.h"
 #include <iostream>
 #include <fstream>
 #include <algorithm>
 #include <numeric>
 #include <hip/hip_runtime.h>
 #include "hip/hip_runtime.h"
 #include ""
 #include <hipcub/hipcub.hpp>
 #include <cub/device/device_radix_sort.cuh>
 #define GLM_FORCE_CUDA
 #include <glm/glm.hpp>
 
 #include <hip/hip_cooperative_groups.h>
 #include <cooperative_groups/reduce.h>
 namespace cg = cooperative_groups;
 
 #include "auxiliary.h"
 #include "forward.h"
 #include "backward.h"
 
 // Helper function to find the next-highest bit of the MSB
 // on the CPU.
 uint32_t getHigherMsb(uint32_t n)
 {
	 uint32_t msb = sizeof(n) * 4;
	 uint32_t step = msb;
	 while (step > 1)
	 {
		 step /= 2;
		 if (n >> msb)
			 msb += step;
		 else
			 msb -= step;
	 }
	 if (n >> msb)
		 msb++;
	 return msb;
 }
 
 // Wrapper method to call auxiliary coarse frustum containment test.
 // Mark all triangles that pass it.
 __global__ void checkFrustum(int P,
	 const float* orig_points,
	 const float* viewmatrix,
	 const float* projmatrix,
	 bool* present)
 {
	 auto idx = cg::this_grid().thread_rank();
	 if (idx >= P)
		 return;
 
	 float3 p_view;
	 present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
 }
 
 // Generates one key/value pair for all Triangle / tile overlaps. 
 // Run once per Triangle (1:N mapping).
 __global__ void duplicateWithKeys(
	 int P,
	 const float2* points_xy,
	 const float* depths,
	 const uint32_t* offsets,
	 const float2* p_image,
	 uint2* rect_min,
	 uint2* rect_max,
	 uint64_t* Triangle_keys_unsorted,
	 uint32_t* Triangle_values_unsorted,
	 int* radii,
	 dim3 grid)
 {
	 auto idx = cg::this_grid().thread_rank();
	 if (idx >= P)
		 return;
 
	 // Generate no key/value pair for invisible triangles
	 if (radii[idx] > 0)
	 {	 uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		 uint2 rect_min_conv = rect_min[idx];
	 	 uint2 rect_max_conv = rect_max[idx];
		 // For each tile that the bounding rect overlaps, emit a 
		 // key/value pair. The key is |  tile ID  |      depth      |,
		 // and the value is the ID of the Triangle. Sorting the values 
		 // with this key yields Triangle IDs in a list, such that they
		 // are first sorted by tile and then by depth. 
		 for (int y = rect_min_conv.y; y < rect_max_conv.y; y++)
		 {
			 for (int x = rect_min_conv.x; x < rect_max_conv.x; x++)
			 {
				 uint64_t key = y * grid.x + x;
				 key <<= 32;
				 key |= *((uint32_t*)&depths[idx]);
				 Triangle_keys_unsorted[off] = key;
				 Triangle_values_unsorted[off] = idx;
				 off++;
			 }
		 }
	 }
 }
 
 // Check keys to see if it is at the start/end of one tile's range in 
 // the full sorted list. If yes, write start/end of this tile. 
 // Run once per instanced (duplicated) Triangle ID.
 __global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
 {
	 auto idx = cg::this_grid().thread_rank();
	 if (idx >= L)
		 return;
 
	 // Read tile ID from key. Update start/end of tile range if at limit.
	 uint64_t key = point_list_keys[idx];
	 uint32_t currtile = key >> 32;
	 if (idx == 0)
		 ranges[currtile].x = 0;
	 else
	 {
		 uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		 if (currtile != prevtile)
		 {
			 ranges[prevtile].y = idx;
			 ranges[currtile].x = idx;
		 }
	 }
	 if (idx == L - 1)
		 ranges[currtile].y = L;
 }
 
 // Mark triangles as visible/invisible, based on view frustum testing
 void CudaRasterizer::Rasterizer::markVisible(
	 int P,
	 float* means3D,
	 float* viewmatrix,
	 float* projmatrix,
	 bool* present)
 {
	 checkFrustum << <(P + 255) / 256, 256 >> > (
		 P,
		 means3D,
		 viewmatrix, projmatrix,
		 present);
 }
 
 CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P, size_t total_nb_points, size_t V)
 {
	 GeometryState geom;
	 obtain(chunk, geom.depths, P, 128);
	 obtain(chunk, geom.clamped, V * 3, 128);
	 obtain(chunk, geom.internal_radii, P, 128);
	 obtain(chunk, geom.means2D, P, 128);
	 obtain(chunk, geom.conic_opacity, P, 128);
	 obtain(chunk, geom.phi_center, P, 128);
	 obtain(chunk, geom.rgb, V * 3, 128);
	 obtain(chunk, geom.tiles_touched, P, 128);
	 hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	 obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	 obtain(chunk, geom.point_offsets, P, 128);
	 obtain(chunk, geom.p_image, total_nb_points, 128);
	 obtain(chunk, geom.indices, total_nb_points, 128);
	 obtain(chunk, geom.offsets, total_nb_points, 128);
	 obtain(chunk, geom.normals, total_nb_points, 128);
	 obtain(chunk, geom.p_w, total_nb_points, 128);
     obtain(chunk, geom.rect_min, P, 128);
     obtain(chunk, geom.rect_max, P, 128);
 
	 return geom;
 }
 
 CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
 {
	 ImageState img;
	 obtain(chunk, img.accum_alpha, N * 3, 128);
	 obtain(chunk, img.n_contrib, N * 2, 128);
	 obtain(chunk, img.ranges, N, 128);
	 return img;
 }
 
 CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
 {
	 BinningState binning;
	 obtain(chunk, binning.point_list, P, 128);
	 obtain(chunk, binning.point_list_unsorted, P, 128);
	 obtain(chunk, binning.point_list_keys, P, 128);
	 obtain(chunk, binning.point_list_keys_unsorted, P, 128);
	 hipcub::DeviceRadixSort::SortPairs(
		 nullptr, binning.sorting_size,
		 binning.point_list_keys_unsorted, binning.point_list_keys,
		 binning.point_list_unsorted, binning.point_list, P);
	 obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	 return binning;
 }
 
 // Forward rendering procedure for differentiable rasterization
 // of triangles.
 int CudaRasterizer::Rasterizer::forward(
	 std::function<char* (size_t)> geometryBuffer,
	 std::function<char* (size_t)> binningBuffer,
	 std::function<char* (size_t)> imageBuffer,
	 const int P, const int V, int D, int M,
	 const float* background,
	 const int width, int height,
	 const float* vertices,
	 const int* triangles_indices,
	 const float* vertex_weights,
	 const float sigma,
	 const int total_nb_points,
	 const float* shs,
	 const float* colors_precomp,
	 float* scaling,
	 const float* viewmatrix,
	 const float* projmatrix,
	 const float* cam_pos,
	 const float tan_fovx, float tan_fovy,
	 const bool prefiltered,
	 float* out_color,
	 float* out_others,
	 float* max_blending,
	 int* radii,
	 bool debug)
 {
	 const float focal_y = height / (2.0f * tan_fovy);
	 const float focal_x = width / (2.0f * tan_fovx);
 
	 size_t chunk_size = required<GeometryState>(P, total_nb_points, V);
	 char* chunkptr = geometryBuffer(chunk_size);
	 GeometryState geomState = GeometryState::fromChunk(chunkptr, P, total_nb_points, V); 
	 
	 if (radii == nullptr)
	 {
		 radii = geomState.internal_radii;
	 }
 
	 dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	 dim3 block(BLOCK_X, BLOCK_Y, 1);
 
	 // Dynamically resize image-based auxiliary buffers during training
	 size_t img_chunk_size = required<ImageState>(width * height);
	 char* img_chunkptr = imageBuffer(img_chunk_size);
	 ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);
 
	 if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	 {
		 throw std::runtime_error("For non-RGB, provide precomputed Triangle colors!");
	 }
 
	 // Run preprocessing per-Triangle (transformation, bounding, conversion of SHs to RGB)
	 CHECK_CUDA(FORWARD::preprocess(
		 P, D, M,
		 vertices,
		 triangles_indices,
		 vertex_weights,
		 sigma,
		 scaling,
		 shs,
		 geomState.clamped,
		 colors_precomp,
		 viewmatrix, projmatrix,
		 (glm::vec3*)cam_pos,
		 width, height,
		 focal_x, focal_y,
		 tan_fovx, tan_fovy,
		 radii,
		 geomState.normals,
		 geomState.offsets,
		 geomState.p_w,
		 geomState.p_image,
		 geomState.indices,
		 geomState.means2D,
		 geomState.depths,
		 geomState.conic_opacity,
		 geomState.phi_center,
		 geomState.rect_min,
		 geomState.rect_max,
		 tile_grid,
		 geomState.tiles_touched,
		 prefiltered
	 ), debug)


	 if (colors_precomp == nullptr)
	{
		// Compute vertex colors in parallel
		FORWARD::computeVertexColors(
			V, D, M,
			vertices,
			shs,
			geomState.clamped,
			geomState.rgb,
			(glm::vec3*)cam_pos
		);
	}
 
	 // Compute prefix sum over full list of touched tile counts by triangles
	 // E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	 CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size, geomState.tiles_touched, geomState.point_offsets, P), debug)
 
	 // Retrieve total number of Triangle instances to launch and resize aux buffers
	 int num_rendered;
	 CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);
 
	 size_t binning_chunk_size = required<BinningState>(num_rendered);
	 char* binning_chunkptr = binningBuffer(binning_chunk_size);
	 BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);
 
	 // For each instance to be rendered, produce adequate [ tile | depth ] key 
	 // and corresponding dublicated Triangle indices to be sorted
	 duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		 P,
		 geomState.means2D,
		 geomState.depths,
		 geomState.point_offsets,
		 geomState.p_image,
		 geomState.rect_min,
		 geomState.rect_max,
		 binningState.point_list_keys_unsorted,
		 binningState.point_list_unsorted,
		 radii,
		 tile_grid)
	 CHECK_CUDA(, debug)
 
	 int bit = getHigherMsb(tile_grid.x * tile_grid.y);
 
	 // Sort complete list of (duplicated) Triangle indices by keys
	 CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		 binningState.list_sorting_space,
		 binningState.sorting_size,
		 binningState.point_list_keys_unsorted, binningState.point_list_keys,
		 binningState.point_list_unsorted, binningState.point_list,
		 num_rendered, 0, 32 + bit), debug)
 
	 CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);
 
	 // Identify start and end of per-tile workloads in sorted list
	 if (num_rendered > 0)
		 identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			 num_rendered,
			 binningState.point_list_keys,
			 imgState.ranges);
	 CHECK_CUDA(, debug)
 
	 // Let each tile blend its range of triangles independently in parallel
	 const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	 CHECK_CUDA(FORWARD::render(
		 tile_grid, block,
		 imgState.ranges,
		 binningState.point_list,
		 width, height,
		 geomState.normals,
		 geomState.offsets,
		 geomState.means2D,
		 triangles_indices,
		 sigma,
		 feature_ptr,
		 geomState.conic_opacity,
		 geomState.depths,
		 geomState.phi_center,
		 geomState.p_image,
		 imgState.accum_alpha,
		 imgState.n_contrib,
		 background,
		 out_color,
		 out_others,
		 max_blending), debug)
 
	 return num_rendered;
 }
 
 // Produce necessary gradients for optimization, corresponding
 // to forward render pass
 void CudaRasterizer::Rasterizer::backward(
	 const int P, const int V, int D, int M, int R,
	 const float* background,
	 const int width, int height,
	 const float* vertices,
	 const int* triangles_indices,
	 const float* vertex_weights,
	 const float sigma,
	 const int total_nb_points,
	 const float* shs,
	 const float* colors_precomp,
	 const float* viewmatrix,
	 const float* projmatrix,
	 const float* campos,
	 const float tan_fovx, float tan_fovy,
	 const int* radii,
	 char* geom_buffer,
	 char* binning_buffer,
	 char* img_buffer,
	 const float* dL_dpix,
	 const float* dL_depths,
	 float* dL_dmeans2D,
	 float* dL_dnormal3D,
	 float* dL_dvertices3D,
	 float* dL_dvertice_weights,
	 float* dL_dnormals,
	 float* dL_doffsets,
	 float* dL_dopacity,
	 float* dL_dcolor,
	 float* dL_dsh,
	 float* dL_dpoints2D,
	 bool debug)
 {
	 GeometryState geomState = GeometryState::fromChunk(geom_buffer, P, total_nb_points, V);
	 BinningState binningState = BinningState::fromChunk(binning_buffer, R);
	 ImageState imgState = ImageState::fromChunk(img_buffer, width * height);
 
	 if (radii == nullptr)
	 {
		 radii = geomState.internal_radii;
	 }
 
	 const float focal_y = height / (2.0f * tan_fovy);
	 const float focal_x = width / (2.0f * tan_fovx);
 
	 const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	 const dim3 block(BLOCK_X, BLOCK_Y, 1);
 
	 // Compute loss gradients w.r.t. 2D mean position, conic matrix,
	 // opacity and RGB of triangles from per-pixel loss gradients.
	 // If we were given precomputed colors and not SHs, use them.
	 const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
	 CHECK_CUDA(BACKWARD::render(
		 tile_grid,
		 block,
		 imgState.ranges,
		 binningState.point_list,
		 width, height,
		 background,
		 sigma,
		 triangles_indices,
		 geomState.normals,
		 geomState.offsets,
		 geomState.conic_opacity,
		 geomState.depths,
		 geomState.means2D,
		 geomState.phi_center,
		 geomState.p_image,
		 color_ptr,
		 imgState.accum_alpha,
		 imgState.n_contrib,
		 dL_dpix,
		 dL_depths,
		 (float2*)dL_dnormals,
		 dL_doffsets,
		 (float3*)dL_dmeans2D,
		 dL_dopacity,
		 dL_dnormal3D,
		 dL_dcolor,
		 dL_dpoints2D), debug)
 

	if (colors_precomp == nullptr) {
		// Compute vertex color gradients in parallel
		CHECK_CUDA(BACKWARD::computeVertexColorGradients(
			V, D, M,
			width, height,
			viewmatrix,
		 	projmatrix,
			geomState.p_w,
			vertices,
			shs,
			geomState.clamped,
			(glm::vec3*)campos,
			dL_dcolor,  // From render output
			dL_dpoints2D,
			(glm::vec3*)dL_dvertices3D,
			dL_dsh
		), debug)
	}


	 // Take care of the rest of preprocessing. Was the precomputed covariance
	 // given to us or a scales/rot pair? If precomputed, pass that. If not,
	 // use the one we computed ourselves.
	 CHECK_CUDA(BACKWARD::preprocess(P, D, M,
		 vertices,
		 triangles_indices,
		 vertex_weights,
		 width, height,
		 radii,
		 shs,
		 geomState.clamped,
		 viewmatrix,
		 projmatrix,
		 geomState.means2D,
		 geomState.p_w,
		 geomState.p_image,
		 geomState.indices,
		 focal_x, focal_y,
		 tan_fovx, tan_fovy,
		 (glm::vec3*)campos,
		 (glm::vec3*)dL_dvertices3D,
		 (float*)dL_dvertice_weights,
		 (float2*)dL_dnormals,
		 dL_doffsets,
		 (float3*)dL_dmeans2D,
		 dL_dopacity,
		 dL_dnormal3D,
		 dL_dcolor,
		 dL_dsh
		), debug)
 }